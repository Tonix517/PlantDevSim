#include "hip/hip_runtime.h"
#include "shadow.h"

#include <assert.h>

MARK *h_ground_shadow;
MARK *d_ground_shadow;

MARK *h_smoothed_shadow;
MARK *d_smoothed_shadow;

const unsigned GroundDim = SCENESIZE * 2/ SpaceGranularity;

void reset_ground_shadow()
{
	assert(h_ground_shadow);
	assert(d_ground_shadow);
	assert(h_smoothed_shadow);
	assert(d_smoothed_shadow);

	memset(h_ground_shadow, 0, sizeof(MARK) * GroundDim * GroundDim);
	hipMemset(d_ground_shadow, 0, sizeof(MARK) * GroundDim * GroundDim);

	memset(h_smoothed_shadow, 0, sizeof(MARK) * pow(SCENESIZE * 2 / SMOOTH_GRAN, 2) );
	hipMemset(d_smoothed_shadow, 0, sizeof(MARK) * pow(SCENESIZE * 2 / SMOOTH_GRAN, 2));
}



///
__device__
int MAX_X_INX = (SCENESIZE * 2) / SpaceGranularity - 1;

__device__
int MAX_Y_INX = (DOMAIN_HEIGHT - SCENEHEIGHT) / SpaceGranularity - 1;

__device__
int MAX_Z_INX = (SCENESIZE * 2) / SpaceGranularity - 1;

__device__
bool isOver(int x, int y, int z)
{
	if( (x < 0 || y < 0 || z < 0) || 
		(x > MAX_X_INX || y > MAX_Y_INX || z > MAX_Z_INX) )
	{
		return true;
	}

	return false;
}

///
__device__
Cell* getCell( unsigned xi, unsigned yi, unsigned zi)
{
	//	haha...
	return _mat + xi * Y_Dim * Z_Dim + yi * Z_Dim + zi;
}

///
///		This piece of code is from Dr. Benes
///		Some changes are made.
///
__device__
float dda_ray_casting(int fromInx[3], int toInx[3])
{

	float fShadowValue = 0;

	int deltaX = toInx[0] - fromInx[0];
	int deltaY = toInx[1] - fromInx[1];
	int deltaZ = toInx[2] - fromInx[2];

	if ( abs(deltaX) >= abs(deltaY) && 
  	     abs(deltaX) >= abs(deltaZ) ) // delta X
	{
		double my = deltaY * 1.f / deltaX;
		double mz = deltaZ * 1.f / deltaX;
		double y = fromInx[1];
		double z = fromInx[2];

		for (int i = fromInx[0]; i < toInx[0]; i++)
		{			
			//	check inx bounds
			if(isOver(i, (int)y, (int)z))
			{
				return fShadowValue;
			}

			fShadowValue += 1 - getCell(i, (int)y, (int)z)->fIllum;
			
			y += my;
			z += mz;
		}
	}
	else if( abs(deltaY) >= abs(deltaX) && 
			 abs(deltaY) >= abs(deltaZ) ) //delta Y
	{
		double mx = deltaX * 1.f / deltaY;
		double mz = deltaZ * 1.f / deltaY;
		double x = fromInx[0];
		double z = fromInx[2];

		for (int j = fromInx[1]; j < toInx[1]; j++)
		{
			//	check inx bounds
			if(isOver((int)x, j, (int)z))
			{
				return fShadowValue;
			}

			fShadowValue += 1 - getCell((int)x, j, (int)z)->fIllum;

			x += mx;
			z += mz;
		}
	}
	else // delta Z
	{
		double mx = deltaX * 1.f / deltaZ;
		double my = deltaY * 1.f / deltaZ;
		double x = fromInx[0];
		double y = fromInx[1];
		for (int k = fromInx[2]; k < toInx[2]; k++)
		{
			//	check inx bounds
			if(isOver((int)x, (int)y, k))
			{
				return fShadowValue;
			}

			fShadowValue += 1 - getCell((int)x, (int)y, k)->fIllum;

			x += mx;
			y += my;
		}
	}

	return 0;
}

__device__
void getCurrentGroundCellInx( unsigned *pxi, unsigned *pyi)
{
	unsigned nAbsTid = blockIdx.x * blockDim.x + threadIdx.x;

	*pxi = nAbsTid % (unsigned)(SCENESIZE * 2 / SpaceGranularity);
	*pyi = nAbsTid / (SCENESIZE * 2 / SpaceGranularity);
}

__device__
void getCurrentSmoothedInx( unsigned *pxi, unsigned *pyi)
{
	unsigned nAbsTid = blockIdx.x * blockDim.x + threadIdx.x;

	*pxi = nAbsTid % (unsigned)(SCENESIZE * 2 / SMOOTH_GRAN);
	*pyi = nAbsTid / (SCENESIZE * 2 / SMOOTH_GRAN);
}

//	Kernel for calculating the shadow
//
__global__
void gpu_calc_shadow( MARK *pBuf, struct Cell *mat)
{
	//
	_mat = mat;

	//
	unsigned xi = 0, zi = 0;
	getCurrentGroundCellInx(&xi, &zi);
	if( (xi + 1) >  GroundDim || (zi + 1) >  GroundDim )
	{
		return;
	}

	//	Get From-Cell index
	int fromInx[3] = {xi, 0, zi};

	int nVerticalCount = (DOMAIN_HEIGHT - SCENEHEIGHT) / SpaceGranularity;
	int nHorizontalCount = (SCENESIZE * 2) / SpaceGranularity;

	//	View Vec
	float sun_vec[3] = SUN_VEC;
	float view_vec[3] = { -sun_vec[0], -sun_vec[1], -sun_vec[2] };
	//assert(view_vec[1] >= 0);

	//	Call To-Cell index
	int toInx[3] = {0};
	toInx[1] = nVerticalCount - 1;
	//	WARNING: the X\Z index may be over the available cube.
	//			 this will be checked in the dda_ray_casting()
	//
	toInx[0] = xi + (nVerticalCount - 1) * (view_vec[0] * 1.f / view_vec[1]);
	toInx[2] = zi + (nVerticalCount - 1) * (view_vec[2] * 1.f / view_vec[1]);
	
	//	Go casting !
	float fShadowValue = dda_ray_casting(fromInx, toInx);
	*(pBuf + zi * nHorizontalCount + xi) = fShadowValue;
}

__global__
void gpu_smooth_shadow( MARK *pBuf, MARK *pSmoothedShadow)
{
	//	Get cell inx first
	unsigned xi = 0, zi = 0;	
	getCurrentSmoothedInx(&xi, &zi);

	const int nSmoothedSize = (SCENESIZE * 2)/ SMOOTH_GRAN;
	if( (xi + 1) >  nSmoothedSize || (zi + 1) >  nSmoothedSize)
	{
		return;
	}

	const int nHorizontalCount = (SCENESIZE * 2) / SpaceGranularity;

	int xi_in_ori = xi * SMOOTH_GRAN / SpaceGranularity;
	int zi_in_ori = zi * SMOOTH_GRAN / SpaceGranularity;
	float deltaX = xi * SMOOTH_GRAN - xi_in_ori * SpaceGranularity;
	float deltaZ = zi * SMOOTH_GRAN - zi_in_ori * SpaceGranularity;

	// Bi-Linear Interpolation
	//  --------
	// | 1 | 2 |
	//  --------
	// | 3 | 4 |
	//  --------
	
	float fUpLeft = 0, fUpRight = 0, fDownLeft = 0, fDownRight = 0; 
	int iFieldNum = 0;

	///	TODO: Yes these if() are stupid enough for CUDA
	///		  there's a better way actually
	///
	if( deltaZ <= (SpaceGranularity / 2))
	{
		if( deltaX <= (SpaceGranularity / 2))	// Field 1
		{
			iFieldNum = 1;

			if(xi_in_ori > 0 && zi_in_ori > 0)
			{
				fUpLeft = *(pBuf + (zi_in_ori - 1) * nHorizontalCount + xi_in_ori - 1);
			}
			if(zi_in_ori > 0)
			{
				fUpRight = *(pBuf + (zi_in_ori - 1) * nHorizontalCount + xi_in_ori);
			}
			if(xi_in_ori > 0)
			{
				fDownLeft =*(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori - 1);
			}			

			fDownRight = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori);
		}
		else	// Field 2
		{
			iFieldNum = 2;

			if(zi_in_ori > 0)
			{
				fUpLeft = *(pBuf + (zi_in_ori - 1) * nHorizontalCount + xi_in_ori);
			}
			if(zi_in_ori > 0 && xi_in_ori < nHorizontalCount)
			{
				fUpRight = *(pBuf + (zi_in_ori - 1) * nHorizontalCount + xi_in_ori + 1);
			}
			
			fDownLeft = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori);
			
			if(xi_in_ori < nHorizontalCount)
			{
				fDownRight = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori + 1);
			}
		}
	}
	else 
	{
		if( deltaX <= (SpaceGranularity / 2) )	//	Field 3
		{
			iFieldNum = 3;

			if(xi_in_ori > 0)
			{
				fUpLeft = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori - 1);
			}
			
			fUpRight = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori);
			
			if(xi_in_ori > 0 && zi_in_ori < nHorizontalCount)
			{
				fDownLeft = *(pBuf + (zi_in_ori + 1) * nHorizontalCount + xi_in_ori - 1);
			}			
			if(zi_in_ori < nHorizontalCount)
			{
				fDownRight = *(pBuf + (zi_in_ori + 1) * nHorizontalCount + xi_in_ori);
			}
		}
		else	//	Field 4
		{
			iFieldNum = 4;

			fUpLeft = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori);

			if(xi_in_ori < nHorizontalCount)
			{
				fUpRight = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori + 1);
			}
			if(zi_in_ori < nHorizontalCount)
			{
				fDownLeft = *(pBuf + (zi_in_ori + 1) * nHorizontalCount + xi_in_ori);
			}
			if(xi_in_ori < nHorizontalCount && zi_in_ori < nHorizontalCount)
			{
				fDownRight = *(pBuf + (zi_in_ori + 1) * nHorizontalCount + xi_in_ori + 1);
			}
		}
	}

	//	All ZERO?
	const float EP = 0.005;
	if( abs(fUpLeft) < EP && 
		abs(fDownRight) < EP && 
		abs(fDownLeft) < EP && 
		abs(fUpRight) < EP )
	{
		*(pSmoothedShadow + zi * nSmoothedSize + xi) = 0;
		return;
	}

	//	Bi-Linear Interpolation
	const float HalfOriCellSize = (SpaceGranularity / 2);
	float fValue = 0;
	float ratioX =   deltaX > HalfOriCellSize ? 
					(deltaX - HalfOriCellSize) / SpaceGranularity : ( (deltaX + HalfOriCellSize) / SpaceGranularity );
	float ratioZ =   deltaZ > HalfOriCellSize ? 
					(deltaZ - HalfOriCellSize) / SpaceGranularity : ( (deltaZ + HalfOriCellSize) / SpaceGranularity );

	fValue = (1 - ratioX) * ( (1 - ratioZ) * fUpLeft + ratioZ * fDownLeft) + 
				   ratioX * ( (1 - ratioZ) * fUpRight + ratioZ * fDownRight) ;

	*(pSmoothedShadow + zi * nSmoothedSize + xi) = fValue;	
	//*(pSmoothedShadow + zi * nSmoothedSize + xi) = *(pBuf + zi_in_ori * nHorizontalCount + xi_in_ori);	
}